
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define PI 3.14159265359f
#define MAX(a,b) (((a)>(b))?(a):(b))
#define p_Nthreads 32

__global__ void jacobi(int N, float * u, float *f, float *unew){
  
  const int i = threadIdx.x + blockIdx.x*blockDim.x + 1; // offset by 1
  const int j = threadIdx.y + blockIdx.y*blockDim.y + 1;

  if (i < N+1 && j < N+1){
    const int Np = (N+2);
    const int id = i + j*(N+2);
    const float ru = -u[id-Np]-u[id+Np]-u[id-1]-u[id+1];
    const float newu = .25 * (f[id] - ru);
    unew[id] = newu;
  }
}

// use all threads
__global__ void reduce(int N2, float *u, float *unew, float *res){

  __shared__ volatile float s_x[p_Nthreads]; // volatile for in-warp smem mods

  const int tid = threadIdx.x;
  const int i = tid + blockIdx.x*(2*blockDim.x);

  s_x[tid] = 0;
  if (i < N2){
    const float unew1 = unew[i];
    const float unew2 = unew[i + blockDim.x];
    const float diff1 = unew1 - u[i];
    const float diff2 = unew2 - u[i + blockDim.x];
    s_x[tid] = diff1*diff1 + diff2*diff2; 

    // update u
    u[i] = unew1;
    u[i + blockDim.x] = unew2;
  }
  __syncthreads();
  
  // stop at s = 64
  for (unsigned int s = blockDim.x/2; s > 32; s /= 2){
    if (tid < s){
      s_x[tid] += s_x[tid+s]; 
    }
    __syncthreads();
  }   

  // manually reduce within a warp
  if (tid < 32){
    s_x[tid] += s_x[tid + 32];
    s_x[tid] += s_x[tid + 16];
    s_x[tid] += s_x[tid + 8];
    s_x[tid] += s_x[tid + 4];
    s_x[tid] += s_x[tid + 2];
    s_x[tid] += s_x[tid + 1];   
  }
  if (tid==0){
    res[blockIdx.x] = s_x[0];
  }
}


int main(int argc, char **argv){
   
  int N = atoi(argv[1]);
  float tol = atof(argv[2]);

  float *u = (float*) calloc((N+2)*(N+2), sizeof(float));
  float *unew = (float*)calloc((N+2)*(N+2),sizeof(float));
  float *f = (float*) calloc((N+2)*(N+2), sizeof(float));
  float h = 2.0/(N+1);
  for (int i = 0; i < N+2; ++i){
    for (int j = 0; j < N+2; ++j){
      const float x = -1.0 + i*h;
      const float y = -1.0 + j*h;
      f[i + j*(N+2)] = sin(PI*x)*sin(PI*y) * h*h;
    }
  } 

  // cuda memory for Jacobi variables
  float *c_u, *c_f, *c_unew;
  hipMalloc(&c_u, (N+2)*(N+2)*sizeof(float));
  hipMalloc(&c_f, (N+2)*(N+2)*sizeof(float));
  hipMalloc(&c_unew, (N+2)*(N+2)*sizeof(float));
  hipMemcpy(c_u,u, (N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(c_f,f, (N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(c_unew,unew,(N+2)*(N+2)*sizeof(float),hipMemcpyHostToDevice);

  // run kernel, copy result back to CPU
  int Nthreads = p_Nthreads; // good if it's a multiple of 32, can't have more than 1024
  int Nblocks = (N + Nthreads-1)/Nthreads; 
  dim3 threadsPerBlock(Nthreads,Nthreads,1);  
  dim3 blocks(Nblocks,Nblocks,1);

  // for reduce kernel
  int Nthreads1D = p_Nthreads; 
  int Nblocks1D = ((N+2)*(N+2) + Nthreads-1)/Nthreads; 
  int halfNblocks1D = (Nblocks1D + 1)/2; 
  dim3 threadsPerBlock1D(Nthreads1D,1,1);  
  dim3 halfblocks1D(halfNblocks1D,1,1);

  // storage for residual
  float *res = (float*) calloc(halfNblocks1D, sizeof(float));
  float *c_res;
  hipMalloc(&c_res, halfNblocks1D*sizeof(float));

  int iter = 0;
  float r2 = 1.;
  while (r2 > tol*tol){

    jacobi <<< blocks, threadsPerBlock >>> (N, c_u, c_f, c_unew);
    reduce <<< halfblocks1D, threadsPerBlock1D >>> ((N+2)*(N+2), c_u, c_unew, c_res);

    // finish block reduction on CPU
    hipMemcpy(res,c_res,halfNblocks1D*sizeof(float),hipMemcpyDeviceToHost);
    r2 = 0.f;
    for (int j = 0; j < halfNblocks1D; ++j){
      r2 += res[j];
    }

    ++iter;
  }
 
  hipMemcpy(u,c_unew,(N+2)*(N+2)*sizeof(float),hipMemcpyDeviceToHost);

  float err = 0.0;
  for (int i = 0; i < (N+2)*(N+2); ++i){
    err = MAX(err,fabs(u[i] - f[i]/(h*h*2.0*PI*PI)));
  }
  
  printf("Max error: %f, r2 = %f, iterations = %d\n", err,r2,iter);

}
  
